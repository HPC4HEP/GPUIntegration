#include "hip/hip_runtime.h"
#include <cstdio>
#include <thread>
#include <future>
#include <functional>
#include <memory>
#include <map>
#include "GPUIntegration/utility.h"
#include "matOps_kernels.cu"


class TaskService{
		class TaskInterface{
			public:
				virtual ~TaskInterface() {};
				virtual std::future<void> launch() =0;
		};
		template<typename Fn> class TaskWrapper;
		template<typename R, typename... Args>
		class TaskWrapper<R(Args...)>: public TaskInterface{
				std::packaged_task<R(Args...)> task_;
				std::thread thread_;
			public:
				TaskWrapper(std::function<R(Args...)>&& f):
										task_(std::forward< std::function<R(Args...)> >(f)) {};
				std::future<void> launch(Args&&... args){
					std::future<void> future= task_.get_future();
					thread_= std::thread(std::move(task_), std::forward<Args>(args)...);
					thread_.detach();
					return future;
				}
		};
		typedef std::unique_ptr<TaskInterface> TaskInterfacePtr;

	public:
		template<typename Fn>
		void set_task(int ID, std::function<Fn>&& f){
			tasks_[ID]= std::move(TaskInterfacePtr(
			          		new TaskWrapper<Fn>(std::forward< std::function<Fn> >(f)) ));
		}
		template<typename... Args>
		std::future<void> launch(int ID, Args&&... args){
			return tasks_.at(ID)->launch(std::forward<Args>(args)...);
		}

	private:
		std::map<int, TaskInterfacePtr> tasks_;
};

class Implementation{
	public:
		virtual void allocate(double*& p, int elemN) =0;
		virtual void execute(const int n, const int times, const double* in, double* out) =0;
		virtual void memfree(double* p) =0;
};
class GPU: public Implementation{
	public:
		void allocate(double*& p, int elemN);
		void execute(const int n, const int times, const double* in, double* out);
		void memfree(double* p);
};
class CPU: public Implementation{
	public:
		void allocate(double*& p, int elemN);
		void execute(const int n, const int times, const double* in, double* out);
		void memfree(double* p);
};

int main()
{
  TaskService taskService;
  Implementation *impl;
	/**Checking presence of GPU**/
  int deviceCount= 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if(error_id == hipSuccess && deviceCount > 0) impl= new GPU;
  else impl= new CPU;
  /*if (error_id == hipErrorNoDevice || deviceCount == 0) impl= new CPU;
  else impl= new GPU;*/
  //impl= new CPU;

	double *in, *out;
	long n;
	n= 20;
	taskService.set_task<void()>(0, [&] {
		impl->allocate(in, n);
		impl->allocate(out, n);
	});
  taskService.launch(0).get();
  
	for(long i=0; i<n; i++) in[i]= 10*sin(PI/100*i);
	for(long i=0; i<n; i++) out[i]= 1;

	taskService.set_task<void()>(1, [&] {
		impl->execute(n, 100, in, out);
	});
  std::future<void> future1= taskService.launch(1);

	future1.get();
	printf("IN:\n");
	for(int i=0; i< n; i++)
		printf("%0.2f\t", in[i]);
	printf("\nOUT:\n");
	for(int i=0; i< n; i++)
		printf("%.0f\t", out[i]);
	printf("\nDONE\n");

	taskService.set_task<void()>(2, [&] {
		impl->memfree(in);
		impl->memfree(out);
	});
	taskService.launch(2).get();
	return 0;
}

void CPU::allocate(double*& p, int elemN)
{
	p= new double[elemN];
}

void CPU::execute(const int n, const int times, const double* in, double* out)
{
	printf("Executing CPU:\n");
	for(int i=0; i<n; i++){
		out[i]= 0;
    for(int t=0; t<times; t++){
      out[i]+= in[i];
    }
	}
}

void CPU::memfree(double* p)
{
	delete(p);
}

void GPU::allocate(double*& p, int elemN)
{
	hipMallocManaged(&p, elemN*sizeof(double));// hipMemAttachHost
}

void GPU::execute(const int n, const int times, const double* in, double* out)
{
	printf("Executing GPU:\n");
	dim3 grid((n-1)/BLOCK_SIZE/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE*BLOCK_SIZE);
	longrunning_kernel<<<grid,block>>>(n, times, in, out);
	hipStreamSynchronize(hipStreamPerThread);
}

void GPU::memfree(double* p)
{
	hipFree(p);
}
