#include "hip/hip_runtime.h"
/// HEMI implementation
#include <cstdio>
#include <thread>
#include <future>
#include "GPUIntegration/utility.h"
#include "matOps_kernels.cu"

#include "hemi/array.h"
#include "hemi/launch.h"

int main()
{
	// m: number of rows
	// n: number of columns
	int m, n;
	// Simple testcase
	m= 10; n= m;

	hemi::Array<float> A(m*n, true), B(m*n, true), C(m*n, true);
	init(A.writeOnlyHostPtr(), B.writeOnlyHostPtr(), m*n);


	std::packaged_task<void()> task([&] (){
		//hemi::ExecutionPolicy policy(ceil(m/BLOCK_SIZE)*ceil(n/BLOCK_SIZE), BLOCK_SIZE);
		hemi::hipLaunchByPtr(matAdd_kernel, m,n, A.readOnlyDevicePtr(), 
		                 B.readOnlyDevicePtr(), C.writeOnlyDevicePtr());
	});
  std::future<void> futureKernel = task.get_future();  
  std::thread(std::move(task)).detach();


	//Output
	/*printf("A:\n");
	show(A.readOnlyHostPtr(), m,n);
	printf("\nB:\n");
	show(B.readOnlyHostPtr(), m,n);
*/
	// Wait for kernel to complete and show result
  futureKernel.get();
	printf("\nC:\n");
	show(C.readOnlyHostPtr(), m,n);
	printf("Done\n\n");
	return 0;
}
