#include "hip/hip_runtime.h"
///Simple CUDA version of matrix addition / Notice the boilerplate
#include <cstdio>
#include <cmath>
#include "matOps_kernels.cu"

#define PI 3.14159265

int main()
{
	float *A, *B, *C;
	// m: number of rows
	// n: number of columns
	int m, n;
	// Simple testcase
	m= 10; n= m;
	A= new float[m*n], B= new float[m*n], C= new float[m*n];
	for (int i=0; i<m*n; i++)
		A[i]= 10*sin(PI/100*i), B[i]= sin(PI/100*i+PI/6)*sin(PI/100*i+PI/6);

	// CUDA boilerplate begin {
	float *dA, *dB, *dC;
	hipMalloc((void **) &dA, m*n*sizeof(float));
	hipMalloc((void **) &dB, m*n*sizeof(float));
	hipMalloc((void **) &dC, m*n*sizeof(float));
	hipMemcpy(dA, A, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, m*n*sizeof(float), hipMemcpyHostToDevice);
	dim3 grid((n-1)/BLOCK_SIZE+1, (m-1)/BLOCK_SIZE+1, 1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	matAdd_kernel<<<grid,block>>>(m, n, dA, dB, dC);
	//(m==n)? matMul_kernel<<<grid,block>>>(m,m,m, dA, dB, dC): exit(1);

	hipMemcpy(C, dC, m*n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dA); hipFree(dB); hipFree(dC);
	// CUDA boilerplate end		}

	//Output
	printf("A:\n");
	for (int y = 0; y < m; y++){
		for (int x = 0; x < n; x++){
			printf("%.2f\t",y,x, A[y*n+x]);
		}
		printf("\n");
	}
	printf("\nB:\n");
	for (int y = 0; y < m; y++){
		for (int x = 0; x < n; x++){
			printf("%.2f\t",y,x, B[y*n+x]);
		}
		printf("\n");
	}
	printf("\nC:\n");
	for (int y = 0; y < m; y++){
		for (int x = 0; x < n; x++){
			printf("%.2f\t",y,x, C[y*n+x]);
		}
		printf("\n");
	}
	free(A); free(B); free(C);
	return 0;
}
