#include <cstdio>
#include "../matOps_kernels.cu"

void allocate(double*& p, int elemN)
{
	hipMallocManaged(&p, elemN*sizeof(double));// hipMemAttachHost
}

void execute(const int n, const int times, const double* in, double* out)
{
	printf("Executing GPU:\n");
	dim3 grid((n-1)/BLOCK_SIZE/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE*BLOCK_SIZE);
	longrunning_kernel<<<grid,block>>>(n, times, in, out);
	hipStreamSynchronize(hipStreamPerThread);
}

void memfree(double* p)
{
	hipFree(p);
}
