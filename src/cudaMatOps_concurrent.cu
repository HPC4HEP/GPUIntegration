/// Concurrent version
#include <cstdio>
#include <thread>
#include <future>
#include "GPUIntegration/utility.h"
#include "matOps_kernels.cu"

int main()
{
	float *A, *B, *C;
	// m: number of rows
	// n: number of columns
	int m, n;
	// Simple testcase
	m= 10; n= m;
	A= new float[m*n], B= new float[m*n], C= new float[m*n];
	init(A, B, m*n);


	std::packaged_task<void()> task([&]{
		float *dA, *dB, *dC;
		hipMalloc((void **) &dA, m*n*sizeof(float));
		hipMalloc((void **) &dB, m*n*sizeof(float));
		hipMalloc((void **) &dC, m*n*sizeof(float));
		hipMemcpy(dA, A, m*n*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dB, B, m*n*sizeof(float), hipMemcpyHostToDevice);
		dim3 grid((n-1)/BLOCK_SIZE+1, (m-1)/BLOCK_SIZE+1);
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		matAdd_kernel<<<grid,block>>>(m, n, dA, dB, dC);
		//(m==n)? matMul_kernel<<<grid,block>>>(m,m,m, dA, dB, dC): exit(1);
		hipMemcpy(C, dC, m*n*sizeof(float), hipMemcpyDeviceToHost);
		hipFree(dA); hipFree(dB); hipFree(dC);
	});
  std::future<void> futureKernel = task.get_future();  
  std::thread(std::move(task)).detach();


	//Output
	printf("A:\n");
	show(A, m,n);
	printf("\nB:\n");
	show(B, m,n);

	// Wait for kernel to complete and show result
  futureKernel.get();
	printf("\nC:\n");
	show(C, m,n);
	printf("Done\n\n");
	free(A); free(B); free(C);
	return 0;
}
